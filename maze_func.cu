#include "hip/hip_runtime.h"
//
// Created by angel on 29/03/2024.
//

#include "maze_func.cuh"
#include "cmath"
#include "ctime"
#include "cstdio"


int return_empty_adjacent(char **maze, Point *adjacent, Point current, int x_max, int y_max){

    Point vector[4];
    int counter = 0;

    if (current.x + 1 < x_max){
        if (maze[current.y][current.x + 1] == EMPTY){
            vector[counter].x = current.x + 1;
            vector[counter].y = current.y;
            counter++;
        }
    }

    if (current.x - 1 > 0){
        if (maze[current.y][current.x - 1] == EMPTY){
            vector[counter].x = current.x - 1;
            vector[counter].y = current.y;
            counter++;
        }
    }

    if (current.y + 1 < y_max){
        if (maze[current.y + 1][current.x] == EMPTY){
            vector[counter].x = current.x;
            vector[counter].y = current.y + 1;
            counter++;
        }
    }

    if (current.y - 1 > 0){
        if (maze[current.y - 1][current.x] == EMPTY){
            vector[counter].x = current.x;
            vector[counter].y = current.y - 1;
            counter++;
        }
    }

    if (counter == 0) return 0;

    for (int i = 0; i < counter; ++i) {
        adjacent[i].x = vector[i].x;
        adjacent[i].y = vector[i].y;
    }

    return counter;
}

int return_adjacent_ways(char **maze, Point current, int x_max, int y_max){
    int counter = 0;

    if (current.x + 1 < x_max){
        if (maze[current.y][current.x + 1] == WAY) counter++;
    }

    if (current.x - 1 > 0){
        if (maze[current.y][current.x - 1] == WAY) counter++;
    }

    if (current.y + 1 < y_max){
        if (maze[current.y + 1][current.x] == WAY) counter++;
    }

    if (current.y - 1 > 0){
        if (maze[current.y - 1][current.x] == WAY) counter++;
    }

    return counter;
}

void wall_filler(char **maze, Point current, int x_max, int y_max){
    int n_ways;
    n_ways = return_adjacent_ways(maze, current, x_max, y_max);
    //printf("(%d, %d)", current.x, current.y);
    //printf(" COUNTS: %d\n", n_ways);
    if (n_ways > 1) maze[current.y][current.x] = WALL;
}

Point path_tracker(char **maze, Point start, int x_max, int y_max, int is_solution){

    int tot_steps;
    if(is_solution == 1){
        tot_steps = (int) (2.5 * x_max);
    }
    else {
        tot_steps = (int) ( sqrt(pow(x_max, 2) + pow(y_max, 2)) * 1.75 );
    }

    //printf("%d\n", tot_steps);
    int step = 0, len, choice;
    Point adjacent[4], current;

    srand(time(nullptr));
    len = return_empty_adjacent(maze, adjacent, start, x_max, y_max);

    while(step < tot_steps){

        if (len != 0){

            choice = rand() % len;

            maze[adjacent[choice].y][adjacent[choice].x] = WAY;
            current = adjacent[choice];
            //printf("POINT: (%d, %d)\nADJACENTS\n", current.x, current.y);
            len = return_empty_adjacent(maze, adjacent, current, x_max, y_max);

            for (int i = 0; i < len; ++i) {
                wall_filler(maze, adjacent[i], x_max, y_max);
            }
            //printf("______________\n\n");

            len = return_empty_adjacent(maze, adjacent, current, x_max, y_max);

        }

        else break;

        step++;
    }

    if (is_solution == 1) maze[current.y][current.x] = START;

    return current;
}

char **maze_init(Point start, Point *solution, int const x_max, int const y_max){

    int counter, len;
    std::vector <Point> vec(x_max * y_max);
    Point neigh[4], current;
    char **maze = (char**) malloc (y_max * sizeof(char*));
    if (maze == nullptr) return nullptr;

    for (int i = 0; i < y_max; ++i) {
        maze[i] = (char*) malloc (x_max * sizeof (char));
        if (maze[i] == nullptr) return nullptr;
    }



    srand(time(nullptr));

    for(int i = 0; i < y_max; i++){
        for (int j = 0; j < x_max; ++j) {
            if(i == 0 || i == y_max -1 || j == 0 || j == x_max - 1){
                maze[i][j] = (i == start.y && j == start.x) ? WAY: WALL;
            }
            else maze[i][j] = EMPTY;
        }
    }

    *solution = path_tracker(maze, start, x_max, y_max, 1);

    do{
        counter = 0;
        for (int i = 0; i < y_max; ++i) {
            for (int j = 0; j < x_max; ++j) {
                if (maze[i][j]  == WAY){
                    current.x = j; current.y = i;
                    len = return_empty_adjacent(maze, neigh, current, x_max, y_max);
                    if (len != 0){
                        vec[counter] = current;
                        counter++;
                    }
                }
            }
        }

        if (counter != 0){
            current = vec[rand() % counter];
            path_tracker(maze, current, x_max, y_max, 0);
        }

    }
    while (counter != 0);

    for (int i = 0; i < y_max; ++i) {
        for (int j = 0; j < x_max; ++j) {
            if (maze[i][j] == EMPTY) maze[i][j] = WALL;
        }
    }

    return maze;
}

void print_maze(char **maze, int x_max, int y_max){
    for (int i = 0; i < y_max; ++i) {
        for (int j = 0; j < x_max; ++j) {
            printf("%c", maze[i][j]);
        }
        printf("\n");
    }
}

int find_ways(char **maze, int x_max, int y_max, Adjacents *adjac, int x, int y){

    int counter = 0;

    if (x + 1 < x_max){
        if (maze[y][x + 1] == WAY || maze[y][x + 1] == START){
            adjac->x[counter] = x + 1;
            adjac->y[counter] = y;
            //adjac->moves[counter] = 'R';
            counter++;
        }
    }

    if (x - 1 >= 0){
        if (maze[y][x - 1] == WAY || maze[y][x - 1] == START){
            adjac->x[counter] = x - 1;
            adjac->y[counter] = y;
            //adjac->moves[counter] = 'L';
            counter++;
        }
    }

    if (y + 1 < y_max){
        if (maze[y + 1][x] == WAY || maze[y + 1][x] == START){
            adjac->x[counter] = x;
            adjac->y[counter] = y + 1;
            //adjac->moves[counter] = 'D';
            counter++;
        }
    }

    if (y - 1 >= 0){
        if (maze[y - 1][x] == WAY || maze[y - 1][x] == START){
            adjac->x[counter] = x;
            adjac->y[counter] = y - 1;
            //adjac->moves[counter] = 'U';
            counter++;
        }
    }

    return counter;
}

int cpu_random_solver(char **maze, int x_max, int y_max, Particles particles, int x_ext, int y_ext){
    int n_ways, rand_choice, n_steps = -1;
    Adjacents *adjac;
    adjac = (Adjacents*) malloc(sizeof(Adjacents));

    //n_steps != NITER
    //true
    while(true){
        n_steps += 1;
        for (int i = 0; i < N; ++i) {
            n_ways = find_ways(maze, x_max, y_max, adjac, particles.x[i], particles.y[i]);
            rand_choice = rand() % n_ways;
            particles.x[i] = adjac->x[rand_choice];
            particles.y[i] = adjac->y[rand_choice];
            //particles.moves[i][n_steps] = adjac->moves[rand_choice];

            //particles.x[i] == x_ext && particles.y[i] == y_ext
            //n_steps == NITER
            if (particles.x[i] == x_ext && particles.y[i] == y_ext){
                free(adjac);
                /*
                printf("PATH:\n");
                for (int j = 0; j < n_steps; ++j) {
                    printf("%c ", particles.moves[i][j]);
                }
                */
                return n_steps + 1;
            }
        }
    }
}

float rand_solver_cpu(const short *h_lin_maze, int x_dim, int y_dim, int x_ext, int y_ext, int n, int *x_array, int *y_array, int max_steps){

    int rand_choice, n_steps = 0, temp, flag = 0;
    clock_t  start, end;

    start = clock();
    while(/*flag == 0 &&*/ n_steps < 1){
        n_steps++;
        for (int i = 0; i < n; ++i){
            for (int j = 0; j < max_steps; j++){
                rand_choice = rand() % (h_lin_maze[11 * y_array[i] * x_dim + 11 * x_array[i] + 2] ) + 1;
                temp = h_lin_maze[11 * y_array[i] * x_dim + 11 * x_array[i] + 2 + 2 * rand_choice - 1];
                y_array[i] = h_lin_maze[11 * y_array[i] * x_dim + 11 * x_array[i] + 2 + 2 * rand_choice];
                x_array[i] = temp;

                //x_vector[i] == x_ext && y_vector[i] == y_ext

                if (x_array[i] == x_ext && y_array[i] == y_ext){
                    flag = 1;
                    end = clock();
                    //return (float) (end - start)/ CLOCKS_PER_SEC;
                }
            }
        }
    }
    end = clock();
    return (float) (end - start)/ CLOCKS_PER_SEC;
}






int distance_evaluate(Point start, Point stop, int dist, int dev){
    //printf("%d\n", dist);
    int man_dist = abs(start.x - stop.x) + abs(start.y - stop.y);
    if(man_dist >= dist - dev && man_dist <= dist + dev) return 1;
    return 0;
}


Point performance_path_tracker(char **maze, Point start, int x_max, int y_max, std::vector<Point>& vec, int *start_found, Point init, Point *solution){

    int tot_steps = (int) ( sqrt(pow(x_max, 2) + pow(y_max, 2)) * 1.75 );

    //printf("%d\n", tot_steps);
    int step = 0, len, choice;
    Point adjacent[4], current;
    current = start;

    srand(time(nullptr));
    len = return_empty_adjacent(maze, adjacent, start, x_max, y_max);

    while(step < tot_steps){

        if (len != 0){
            if (len == 1){
                vec.erase(std::remove(vec.begin(), vec.end(), current), vec.end());
            }


            choice = rand() % len;

            if (*start_found == 0){
                if (distance_evaluate(init, adjacent[choice],1.75 * x_max , 5) == 1){
                    *start_found = 1;
                    maze[adjacent[choice].y][adjacent[choice].x] = START;
                    *solution = adjacent[choice];
                    //printf("DISTANCE: %d\n", abs(init.x - adjacent[choice].x) + abs(init.y - adjacent[choice].y));
                    break;
                }
            }

            maze[adjacent[choice].y][adjacent[choice].x] = WAY;
            current = adjacent[choice];
            //printf("POINT: (%d, %d)\nADJACENTS\n", current.x, current.y);
            len = return_empty_adjacent(maze, adjacent, current, x_max, y_max);

            for (int i = 0; i < len; ++i) {
                wall_filler(maze, adjacent[i], x_max, y_max);
            }
            //printf("______________\n\n");

            vec.push_back(current);
            len = return_empty_adjacent(maze, adjacent, current, x_max, y_max);

        }

        else{
            vec.erase(std::remove(vec.begin(), vec.end(), current), vec.end());
            break;
        }

        step++;
    }

    return current;
}

char **performance_maze_init(Point start, Point *solution, int x_max, int y_max){

    int counter = 0, len;
    std::vector <Point> vec(1);
    vec[0] = start;

    int *start_found;
    start_found = (int*) malloc(sizeof(int));
    *start_found = 0;

    Point neigh[4], current = start;
    char **maze = (char**) malloc (y_max * sizeof(char*));
    if (maze == nullptr) return nullptr;

    srand(time(nullptr));

    for (int i = 0; i < y_max; ++i) {
        maze[i] = (char*) malloc (x_max * sizeof (char));
        if (maze[i] == nullptr) return nullptr;

        for (int j = 0; j < x_max; ++j) {
            if(i == 0 || i == y_max -1 || j == 0 || j == x_max - 1){
                maze[i][j] = (i == start.y && j == start.x) ? WAY : WALL;
            }
            else{
                maze[i][j] = EMPTY;
            }
        }
    }

    while (vec.size() != 0) {
        current = vec[rand() % (int) vec.size()];
        performance_path_tracker(maze, current, x_max, y_max, vec, start_found, start, solution);
    }

    free(start_found);
    return maze;
}




